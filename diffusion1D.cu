#include "hip/hip_runtime.h"


#include <iostream>
#include <math.h>

using namespace std;

int main(){
    int nx = 20;
    int nt = 50;
    float dt = 0.01;
    float vis = 0.1; // diffusion parameter if v>0: physical diffusion, if v<0: represents an exponentially gorowing phenomena
    float dx = 2.0 / (nx - 1);

    float u[nx];

    //populate IC array
    for(int i = 0; i < nx; i++){
        if(i >= 0.5 && i <= 1){
            u[i] = 2;
        }
        else{
            u[i] = 1;
        }
    }

    //print the initial condition
    printf("Initial u:\n");
    for(int i = 0; i < nx; i++){
        cout << i << ": " << u[i] << endl;
    }

    float un[nx];
    for(int it = 1; it < nt; it++){
        for(int i = 0; i < nx; i++){
            un[i] = u[i];
        }
        for(int i = 1; i < nx-1; i++){
            u[i] = un[i] + vis*dt/dx*(un[i+1] - 2*un[i] + un[i-1]);
        }
    }

    printf("Final u:\n");
    for(int i = 0; i < nx; i++){
        cout << i << ": " << un[i] << endl;
    }
    return 0; 
}