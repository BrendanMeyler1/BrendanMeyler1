//1D convection NSE also known as "invisid Burgers' equation"
// it can generate shock waves
// ie can generate discontinuities in the solution from smooth IC
// similar to shock creation at supersonic speeds
// using the forward and backward euler method 

#include "hip/hip_runtime.h"


#include <iostream>
#include <math.h>

using namespace std;

int main(){
    int nx = 20;
    int nt = 50;
    float dt = 0.01;
    float dx = 2.0 / (nx - 1);

    float u[nx];
    //populate IC array
    for(int i = 0; i < nx; i++){
        if(i >= 0.5 && i <= 1){
            u[i] = 2;
        }
        else{
            u[i] = 1;
        }
    }

    //print the initial condition
    printf("Initial u:\n");
    for(int i = 0; i < nx; i++){
        cout << i << ": " << u[i] << endl;
    }

    float un[nx];
    for(int it = 1; it < nt; it++){
        for(int i = 0; i < nx; i++){
            un[i] = u[i];
        }
        for(int i = 1; i < nx; i++){
            u[i] = un[i] - un[i]*dt/dx*(un[i] - un[i-1]);
        }
    }

    printf("Final u:\n");
    for(int i = 0; i < nx; i++){
        cout << i << ": " << un[i] << endl;
    }
    return 0;

}