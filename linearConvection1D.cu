#include "hip/hip_runtime.h"


#include <iostream>
#include <math.h>

using namespace std;

int main(){
    int nx = 30; // number of grid points (domain size)
    int nt = 60; // total time
    float c = 1; //convective speed
    float dx = 2.0 / (nx - 1); // distance between grid points
    float dt = 0.0025; //time step size
    float u[nx]; //intialize the array to hold IC

    //populate IC array
    for(int i = 0; i < nx; i++){ 
        if(i >= 0.5 && i <= 3){
            u[i] = 2;
        }else{
            u[i] = 1;
        }
    }
    //print the initial condition
    printf("Initial u:\n");
    for(int i = 0; i < nx; i++){
        cout << i << ": " << u[i] << endl;
    }

    // loop to update the array
    float un[nx];
    for(int i = 0; i < nt; i++){
        //cout << "Array update" << endl;
        for(int j = 0; j < nx; j++){
            un[j] = u[j];
            //cout << un[j] << endl;
        }
        for(int j = 2; j < nx-1; j++){
            u[j] = un[j] - c * dt / dx * (un[j] - un[j - 1]);
            //cout << u[j] << endl;
        }
    }
    printf("Final u:\n");
    for(int i = 0; i < nx; i++){
        cout << i << ": " << un[i] << endl;
    }
    return 0;
}